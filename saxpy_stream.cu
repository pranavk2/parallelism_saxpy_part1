#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "CycleTimer.h"
#include "saxpy.h"
__global__ void
saxpy_kernel(int N, float alpha, float* x, float* y, float* result) {
	// compute overall index from position of thread in current block,
	// and given the block we are in
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N)
		result[index] = alpha * x[index] + y[index];
}
static inline
int getBlocks(long working_set_size, int threadsPerBlock) {
	// TODO: implement and use this interface if necessary  
}
void 
getArrays(int size, float **xarray, float **yarray, float **resultarray) {
	// TODO: implement and use this interface if necessary  
}
void 
freeArrays(float *xarray, float *yarray, float *resultarray) {
	// TODO: implement and use this interface if necessary  
}
void
saxpyCuda(long total_elems, float alpha, float* xarray, float* yarray, float* resultarray, int partitions) {
	const int threadsPerBlock = 512; // change this if necessary
	// printf("partitions is %d\n ", partitions);
	float *device_x;
	float *device_y;
	float *device_result;
	//
	// TODO: allocate device memory buffers on the GPU using
	// hipMalloc.  The started code issues warnings on build because
	// these buffers are used in the call to saxpy_kernel below
	// without being initialized.
	//
	hipMalloc(&device_x,total_elems*sizeof(float));
	hipMalloc(&device_y,total_elems*sizeof(float));
	hipMalloc(&device_result,total_elems*sizeof(float));
	hipStream_t Stream[partitions];
	hipError_t Result[partitions];
	int streamSize=total_elems/partitions;
	int numBlocks=(total_elems/partitions+threadsPerBlock-1)/threadsPerBlock;
	int offset;
	// start timing after allocation of device memory.
	double startTime = CycleTimer::currentSeconds();
	//double startH2D,endH2D,startD2H,endD2H,timeKernel;
	for (int i=0; i<partitions; i++) {   
		offset=i*streamSize;
		//
		// TODO: copy input arrays to the GPU using hipMemcpy
		//
		//startH2D=CycleTimer::currentSeconds();
		Result[i]=hipStreamCreate(&Stream[i]);
		hipMemcpyAsync(&(device_x[offset]),&(xarray[offset]),streamSize*sizeof(float),hipMemcpyHostToDevice,Stream[i]);
		hipMemcpyAsync(&(device_y[offset]),&(yarray[offset]),streamSize*sizeof(float),hipMemcpyHostToDevice,Stream[i]);
		//endH2D=CycleTimer::currentSeconds();
		//
		// TODO: insert time here to begin timing only the kernel
		//
		//double startGPUtime=CycleTimer::currentSeconds();
		// compute number of blocks and threads per block
		// run saxpy_kernel on the GPU
		saxpy_kernel<<<numBlocks, threadsPerBlock,0,Stream[i]>>>(streamSize,alpha,device_x+offset,device_y+offset,device_result+offset);
		//
		// TODO: insert timer here to time only the kernel.  Since the
		// kernel will run asynchronously with the calling CPU thread, you
		// need to call hipDeviceSynchronize() before your timer to
		// ensure the kernel running on the GPU has completed.  (Otherwise
		// you will incorrectly observe that almost no time elapses!)
		//
		//hipDeviceSynchronize();
		//double endGPUtime=CycleTimer::currentSeconds();
		//timeKernel=endGPUtime-startGPUtime;
		//printf("kernel %d time is %f\n",i,timeKernel);        
		hipError_t errCode = hipPeekAtLastError();
		if (errCode != hipSuccess) {
			fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
		}

		//
		// TODO: copy result from GPU using hipMemcpy
		//
		// startD2H=CycleTimer::currentSeconds();
		hipMemcpyAsync(&(resultarray[offset]),&(device_result[offset]),streamSize*sizeof(float),hipMemcpyDeviceToHost,Stream[i]);
		//endD2H=CycleTimer::currentSeconds(); 
		Result[i]=hipStreamDestroy(Stream[i]);  
	}
	hipDeviceSynchronize();
	// end timing after result has been copied back into host memory.
	// The time elapsed between startTime and endTime is the total
	// time to copy data to the GPU, run the kernel, and copy the
	// result back to the CPU

	double endTime = CycleTimer::currentSeconds();
	double overallDuration = endTime - startTime;
	totalTimeAvg+=overallDuration;
	//timeKernelAvg+=timeKernel;
	//timeCopyH2DAvg+=(endH2D-startH2D);
	//timeCopyD2HAvg+=(endD2H-startD2H);

	//
	// TODO free memory buffers on the GPU
	//
	hipFree(device_x);
	hipFree(device_y);
	hipFree(device_result);
}
void
printCudaInfo() {
	// for fun, just print out some stats on the machine
	int deviceCount = 0;
	hipError_t err = hipGetDeviceCount(&deviceCount);
	printf("---------------------------------------------------------\n");
	printf("Found %d CUDA devices\n", deviceCount);
	for (int i=0; i<deviceCount; i++) {
		hipDeviceProp_t deviceProps;
		hipGetDeviceProperties(&deviceProps, i);
		printf("Device %d: %s\n", i, deviceProps.name);
		printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
		printf("   Global mem: %.0f MB\n",
				static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
		printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
	}
	printf("---------------------------------------------------------\n");
}

